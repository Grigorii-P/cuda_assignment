#include "hip/hip_runtime.h"
/**
 * 
 * Matrix Multiplication - CUDA for GPUs
 *
 * CS3210
 *
 **/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

// Thread block size
#define BLOCK_SIZE 32
#define STRIDE BLOCK_SIZE

int size;

typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;


__device__ float GetElement(const Matrix A, int row, int col) {
    return A.elements[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value) {
    A.elements[row * A.stride + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
Matrix Asub;
Asub.width = BLOCK_SIZE;
Asub.height = BLOCK_SIZE;
Asub.stride = A.stride;
Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
return Asub;
}


long long wall_clock_time()
{
#ifdef __linux__
	struct timespec tp;
	clock_gettime(CLOCK_REALTIME, &tp);
	return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

void allocate_matrix(Matrix* m)
{

	m->elements = (float*)malloc(size * size * sizeof(float));


	// int i;
	// m->elements = (float**)malloc(sizeof(float*) * size);
	// if (m->elements == NULL)
	// {
	// 	fprintf(stderr, "Out of memory\n");
	// 	exit(1);
	// }
	
	// // allocate an array for each row of the matrix
	// for (i = 0; i < size; i++)
	// {
	// 	m->elements[i] = (float*)malloc(sizeof(float) * size);
	// 	if (m->elements[i] == NULL)
	// 	{
	// 		fprintf(stderr, "Out of memory\n");
	// 		exit(1);
	// 	}
	// }
}

/**
 * Free the memory allocated for a matrix.
 **/
void free_matrix(Matrix* m) {
	int i;
	for (i = 0; i < size; i++)
		hipFree(m->elements[i]);
	hipFree(m->elements);
}

/**
 * Initializes the elements of the matrix with
 * random values between 0 and 9
 **/
void init_matrix(Matrix m)
{
	m.stride = STRIDE;
	for (i = 0; i < size*size; i++) {
		m.elements[i] = rand() % 10;
	}

	// int i, j;
	
	// for (i = 0; i < size; i++)
	// 	for (j = 0; j < size; j++)
	// 	{
	// 		m.elements[i][j] = rand() % 10;
	// 	}
	
}


__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.stride = A.width;
    d_A.width = d_A.stride; 
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.stride = B.width;
    d_B.width = d_B.stride; 
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    
    // Allocate C in device memory
    Matrix d_C;
    d_C.stride = C.width;
    d_C.width = d_C.stride; 
    d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);
    
    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    
    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}


// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;
    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;
    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);
        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);
        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
	}	
    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}


void work()
{
	Matrix a, b, result1, result2;
	long long before, after;
	int correct, i, j, dim;
	hipError_t rc;

	// Allocate memory for matrices
	allocate_matrix(&a);
	allocate_matrix(&b);
	allocate_matrix(&result1);
	allocate_matrix(&result2);	

	// Initialize matrix elements
	init_matrix(a);
	init_matrix(b);

	// Perform CUDA matrix  multiplication
	before = wall_clock_time();
	MatMul(a, b, result2);
	hipDeviceSynchronize();
	after = wall_clock_time();
	fprintf(stderr, "Matrix multiplication on GPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

	// was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

	free_matrix(&a);
	free_matrix(&b);
	free_matrix(&result1);
	free_matrix(&result2);
}


int main(int argc, char ** argv)
{
	srand(0); 

	printf("Usage: %s <size>\n", argv[0]);
    
	if (argc >= 2)
		size = atoi(argv[1]);
	else
		size = 1024;
		
	fprintf(stderr,"Sequential matrix multiplication of size %d\n", size);
    
	// Multiply the matrices
	work();

	return 0;
}
